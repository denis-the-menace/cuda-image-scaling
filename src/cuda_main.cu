#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <time.h>

#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION

#include "../include/stb_image.h"
#include "../include/stb_image_write.h"
#define CHANNEL_NUM 1

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

typedef struct
{
  unsigned char red, green, blue;
} Pixel;

__global__ void bilinear_downscaling_kernel(uint8_t *image, Pixel *pixels, int bpp,
                                            int width, int height, int new_width,
                                            int new_height, float scale_x,
                                            float scale_y)
{
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x < new_width && y < new_height)
  {
    Pixel pixel = {255, 255, 255};

    float src_x = x * scale_x;
    float src_y = y * scale_y;

    int x1 = (int)floorf(src_x);
    int y1 = (int)floorf(src_y);
    int x2 = x1 + 1;
    int y2 = y1 + 1;

    float w1 = (x2 - src_x) * (y2 - src_y);
    float w2 = (src_x - x1) * (y2 - src_y);
    float w3 = (x2 - src_x) * (src_y - y1);
    float w4 = (src_x - x1) * (src_y - y1);

    int index1 = (y1 * width + x1) * bpp;
    int index2 = (y1 * width + x2) * bpp;
    int index3 = (y2 * width + x1) * bpp;
    int index4 = (y2 * width + x2) * bpp;

    Pixel pixel1 = {image[index1], image[index1 + 1], image[index1 + 2]};
    Pixel pixel2 = {image[index2], image[index2 + 1], image[index2 + 2]};
    Pixel pixel3 = {image[index3], image[index3 + 1], image[index3 + 2]};
    Pixel pixel4 = {image[index4], image[index4 + 1], image[index4 + 2]};

    pixel.red = (unsigned char)(w1 * pixel1.red + w2 * pixel2.red +
                                w3 * pixel3.red + w4 * pixel4.red);
    pixel.green = (unsigned char)(w1 * pixel1.green + w2 * pixel2.green +
                                  w3 * pixel3.green + w4 * pixel4.green);
    pixel.blue = (unsigned char)(w1 * pixel1.blue + w2 * pixel2.blue +
                                 w3 * pixel3.blue + w4 * pixel4.blue);

    pixels[y * new_width + x] = pixel;
  }
}

void bilinear_downscaling(uint8_t *image, Pixel *pixels, int bpp, int width,
                          int height, int new_width, int new_height,
                          float scale_x, float scale_y)
{

  uint8_t *d_image;
  Pixel *d_pixels;

  hipMalloc((void **)&d_image, width * height * bpp);
  hipMalloc((void **)&d_pixels, new_width * new_height * sizeof(Pixel));

  hipMemcpy(d_image, image, width * height * bpp, hipMemcpyHostToDevice);

  dim3 blockSize(16, 16);
  dim3 gridSize((new_width + blockSize.x - 1) / blockSize.x,
                (new_height + blockSize.y - 1) / blockSize.y);

  bilinear_downscaling_kernel<<<gridSize, blockSize>>>(d_image, d_pixels, bpp,
                                                       width, height,
                                                       new_width, new_height,
                                                       scale_x, scale_y);

  hipMemcpy(pixels, d_pixels, new_width * new_height * sizeof(Pixel),
             hipMemcpyDeviceToHost);

  hipFree(d_image);
  hipFree(d_pixels);
}

int main(int argc, char *argv[])
{
  int width, height, bpp, new_width, new_height;
  float scale_x, scale_y;
  clock_t start, end;

  uint8_t *image = stbi_load(argv[1], &width, &height, &bpp, 0);

  new_width = width / 2;
  new_height = height / 2;

  scale_x = width / (float)new_width;
  scale_y = height / (float)new_height;

  printf("New width: %d, New height: %d\n", new_width, new_height);
  printf("Scale factor for x: %d, Scale factor for y: %d\n", (int)scale_x, (int)scale_y);

  Pixel *pixels = (Pixel *)malloc(new_width * new_height * sizeof(Pixel));

  start = clock();
  bilinear_downscaling(image, pixels, bpp, width, height, new_width, new_height,
                       scale_x, scale_y);
  stbi_write_jpg(argv[2], new_width, new_height, 3, pixels, 100);
  end = clock();

  double duration = ((double)end - start)/CLOCKS_PER_SEC;

  printf("Elapsed time : %lf for CUDA GPU\n", duration);

  stbi_image_free(image);
  free(pixels);
  return 0;
}
